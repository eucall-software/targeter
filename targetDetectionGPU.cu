#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <stdlib.h>
#include "math.h"
#include "targetDetectionGPU.cuh"
#include "globals.h"

texture<int, hipTextureType2D, hipReadModeElementType> cuda_texture_image;

bool Check_CUDA_Device()
{
	int devID = 0;
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	error = hipGetDeviceProperties(&deviceProp, devID);

	int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		DBOUT("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
		DBOUT("\nFAILED\n");
	}

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		DBOUT("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		return false;
	}
	else
	{
		DBOUT("GPU Device " << devID << " - " << deviceProp.name << " with compute capability major=" << deviceProp.major << " minor=" << deviceProp.minor << std::endl);

		const int kb = 1024;
		DBOUT("  Shared memory: " << (int)deviceProp.sharedMemPerBlock / kb << " MB" << std::endl);
		DBOUT("  Constant memory: " << (int)deviceProp.totalConstMem / kb << " MB" << std::endl);
		DBOUT("  Block registers: " << deviceProp.regsPerBlock << std::endl);
		DBOUT("  Warp size:        " << deviceProp.warpSize << std::endl);
		DBOUT("  Threads per block: " << deviceProp.maxThreadsPerBlock << std::endl);
		DBOUT("  Max block dimensions: " << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << std::endl);
		DBOUT("  Max grid dimensions:  " << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << std::endl);

		int driverVersion = 0, runtimeVersion = 0;

		if (devID == 0) {
			// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
			if (deviceProp.major == 9999 && deviceProp.minor == 9999)
				printf("There is no device supporting CUDA.\n");
			else if (deviceCount == 1)
				printf("There is 1 device supporting CUDA\n");
			else
				printf("There are %d devices supporting CUDA\n", deviceCount);
		}

		printf("\nDevice %d: \"%s\"\n", devID, deviceProp.name);

		typedef struct
		{
			int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
			int Cores;
		} sSMtoCores;

		sSMtoCores nGpuArchCoresPerSM[] =
		{
			{ 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
			{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
			{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
			{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
			{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
			{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
			{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
			{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
			{ -1, -1 }
		};


#if CUDART_VERSION >= 2020
		// Console log
		hipDriverGetVersion(&driverVersion);
		printf("  CUDA Driver Version:                           %d.%d\n", driverVersion / 1000, driverVersion % 100);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Runtime Version:                          %d.%d\n", runtimeVersion / 1000, runtimeVersion % 100);
#endif
		printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
		printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);

		printf("  Total amount of global memory:                 %u bytes\n", deviceProp.totalGlobalMem);
#if CUDART_VERSION >= 2000
		printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
		printf("  Number of cores:                               %d\n", nGpuArchCoresPerSM[deviceProp.major].SM * deviceProp.multiProcessorCount);
#endif
		printf("  Total amount of constant memory:               %u bytes\n", deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %u bytes\n", deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		printf("  Warp size:                                     %d\n", deviceProp.warpSize);
		printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %u bytes\n", deviceProp.memPitch);
		printf("  Texture alignment:                             %u bytes\n", deviceProp.textureAlignment);
		printf("  Clock rate:                                    %.2f GHz\n", deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
		printf("  Concurrent copy and execution:                 %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
#endif
#if CUDART_VERSION >= 2020
		printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated:                                    %s\n", deviceProp.integrated ? "Yes" : "No");
		printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
		printf("  Compute mode:                                  %s\n", deviceProp.computeMode == hipComputeModeDefault ?
			"Default (multiple host threads can use this device simultaneously)" :
			deviceProp.computeMode == hipComputeModeExclusive ?
			"Exclusive (only one host thread at a time can use this device)" :
			deviceProp.computeMode == hipComputeModeProhibited ?
			"Prohibited (no host thread can use this device)" :
			"Unknown");
#endif
#if CUDART_VERSION >= 3000
		printf("  Concurrent kernel execution:                   %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
#endif
#if CUDART_VERSION >= 3010
		printf("  Device has ECC support enabled:                %s\n", deviceProp.ECCEnabled ? "Yes" : "No");
#endif
		// csv masterlog info
		// *****************************
		// exe and CUDA driver name 
		printf("\n");
		std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
		char cTemp[10];

		// driver version
		sProfileString += ", CUDA Driver Version = ";

		sprintf(cTemp, "%d.%d", driverVersion / 1000, driverVersion % 100);

		sProfileString += cTemp;

		// Runtime version
		sProfileString += ", CUDA Runtime Version = ";

		sprintf(cTemp, "%d.%d", runtimeVersion / 1000, runtimeVersion % 100);

		sProfileString += cTemp;

		// Device count      
		sProfileString += ", NumDevs = ";

		sprintf(cTemp, "%d", deviceCount);

		sProfileString += cTemp;

		// First 2 device names, if any
		for (devID = 0; devID < ((deviceCount > 2) ? 2 : deviceCount); ++devID)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, devID);
			sProfileString += ", Device = ";
			sProfileString += deviceProp.name;
		}
		sProfileString += "\n";

		printf("%s", sProfileString.c_str());
	}
	return true;
}

#define checkCudaErrors(err) { __checkCudaErrors((err), __FILE__, __LINE__); }
inline void __checkCudaErrors(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		const char* es = hipGetErrorString(err);

		DBOUT("file=" << (int)file << " line= "  << line << " CUDA Runtime API error (" << (int)err << ")" <<  es << std::endl);
	}
}

cudaScore::cudaScore(int w_d, int h_d, int CoocSize)
{
	// allocate image
	hipMalloc(&cuda_intensity_image, w_d * h_d * sizeof(int));

	// allocate target image
	hipMalloc(&cuda_target_cooc, CoocSize * sizeof(float));

	// allocate score image
	hipMalloc(&cuda_score_image, w_d * h_d * sizeof(float));

	hipMalloc(&cuda_score_count_image, w_d * h_d * sizeof(int));
}

cudaScore::~cudaScore()
{
	hipFree(cuda_intensity_image);
	hipFree(cuda_target_cooc);
	hipFree(cuda_score_image);
	hipFree(cuda_score_count_image);
}


__global__
void mycudaMemsetInt(int* image, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		image[i] = 0;
	}
}

__global__
void mycudaMemsetFloat(float* image, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		image[i] = 0.0;
	}
}

/**
*
*  Gets coocurrance matrix for image region for grayscale image
*
* @author    David Watts
* @since     2017/03/07
*
* FullName   getCoocMatrixGrayGPU
* Qualifier
* @param     int* & m
* @param     int * pMask
* @param     int startX
* @param     int startY
* @param     int regionWidth
* @param     int regionHeight
* @param     float * coMatrixF
* @param     int NoClusters
* @param     int maxD
* @return    float
* Access     public
*/
__device__
void getCoocMatrixGrayGPU(int image_ind_x, int image_ind_y, int width, int height, 
						int regionWidth, int regionHeight, int* coMatrixRegion, int coDIMX, int coDIMY, int coDIMZ)	
{
	int local_sum = 0;
	int pixel_index = image_ind_x + image_ind_y*width;

	if (threadIdx.x < regionWidth && threadIdx.y < regionHeight)
	{
		int c1 = tex2D(cuda_texture_image, image_ind_x, image_ind_y); // pImage[threadIdx.x + threadIdx.y * regionWidth];

		if (c1 >= 0)
		{
			int nextIndex = 1 + threadIdx.x + threadIdx.y*regionWidth;

			int nextJ = nextIndex / regionWidth;
			int nextI = nextIndex % regionWidth;

			//printf("rw=%d rh=%d\n", regionWidth, regionHeight);

			// check against all others
			for (int y = nextJ; y < regionHeight; y++)
				for (int x = nextI; x < regionWidth; x++)
				{
					if (image_ind_x + x < width && image_ind_y + y < height)
					{
						int c2 = tex2D(cuda_texture_image, image_ind_x + x, image_ind_y + y);  //pImage[x + y*regionWidth];

						if (c2 >= 0)
						{
							int ind = c1 + coDIMX * (c2 + coDIMY * MAX(x, y));

							if (ind < coDIMX*coDIMY*coDIMZ)
							{
								atomicAdd(&coMatrixRegion[ind], 1);
								atomicAdd(&coMatrixRegion[0], 1);	// not needed, can be precomputed as n*(n-1)/2
							}
						}
					}
				}
		}
	}
}

/**
*
*  Gets target score image based on matching image regions with training coccurrance mattrix (coMatrixTestF)
*
* @author    David Watts
* @since     2017/03/07
*
* FullName   findTargets
* Qualifier
* @param     int* & detectionImage
* @param     float * coMatrixTestF
* @param     int NoClusters
* @param     int maxD
* @return    float*
* Access     public
*/
__global__
void scoreTargetsGPU(int w, int h, int regionWidth, int regionHeight, int increment_W, int increment_H,
					float* coMatrixTarget, float* scoreImage, int* countImage, int coDIMX, int coDIMY, 
					int coDIMZ, int* ct, bool bCrossEntropy)
{
	__shared__ int shared_count;
	__shared__ float shared_score;

	extern __shared__ int sh[];
	int* sharedCoocRegion = sh;
	//int* sharedCoocRegion = (int*)sh + regionWidth*regionHeight;

	// index into image and sub tile
	int image_ind_x = blockIdx.x*increment_W + threadIdx.x;
	int image_ind_y = blockIdx.y*increment_H + threadIdx.y;

	// zero shared memory
	int tid = threadIdx.y*blockDim.x + threadIdx.x;			// get linear thread index
	int cN = coDIMX*coDIMY*coDIMZ;
	int blockSize = blockDim.x * blockDim.y;

	// in the case of sharedCooc having bigger dimensions than the block size
	// zero remaining part of memory
	if (tid == 0)
	{
		shared_count = 0;
		shared_score = 0.0;
	}

	// zero co-occurrence matrix using parralel threads
	for(int i=tid; i<cN; i += blockSize)
		sharedCoocRegion[i] = 0;
/*
	// copy image region into into shared tile memory
	if (image_ind_x < w && image_ind_y < h)
		sharedTile[threadIdx.x + threadIdx.y*regionWidth] = tex2D(cuda_texture_image, image_ind_x, image_ind_y);
	else
		sharedTile[threadIdx.x + threadIdx.y*regionWidth] = -1;
*/
	__syncthreads();

	// create the coocurrence matrices

	// last data point has nothing to check against
	if (threadIdx.x != regionHeight - 1 && threadIdx.y != regionHeight - 1)
	{
		// takes pixel value compares with neighbours
		getCoocMatrixGrayGPU(image_ind_x, image_ind_y, w, h, regionWidth, regionHeight, sharedCoocRegion, coDIMX, coDIMY, coDIMZ);
	}

	__syncthreads();

	// loop though coocurance matrices and get score
	float local_score = 0.0;

	for (int i = tid; i < cN; i += blockSize)
	{
		float score_test = coMatrixTarget[i];
		float score_train = float(sharedCoocRegion[i]) / float(sharedCoocRegion[0]);
		local_score += sqrt(score_test*score_train);
	}

	if (local_score>0)
		atomicAdd(&shared_score, local_score);

	__syncthreads();

	// label image region with score

	if (image_ind_x < w && image_ind_y < h)
	{
		atomicAdd(&scoreImage[image_ind_x + image_ind_y*w], shared_score);

		//scoreImage[image_ind_x + image_ind_y*w] = threadIdx.x;

		atomicAdd(&countImage[image_ind_x + image_ind_y*w], 1);

		//printf("%f ", scoreImage[image_ind_x + image_ind_y*w]);
	}
	
	//////////////////////////////// sum check /////////////////////////////////////
	atomicAdd(ct, 1);
}


__global__
void cudaEnergy(int* image, int w, int h, double* dev_score)
{
	int i = (blockIdx.y * blockDim.y + threadIdx.y);
	int j = (blockIdx.x * blockDim.x + threadIdx.x);

	// Test to see if we're testing a valid pixel
	if (i < w && j < h)
	{
		int diff1 = 0, diff2 = 0;

		int val1 = image[i + j*w];

		if ((i + 1) < w)
			diff1 = val1 - image[(i + 1) + j*w];

		if ((j + 1) < h)
			diff2 = val1 - image[i + (j + 1)*w];
		
		atomicAdd(dev_score, (double)diff1*diff1 + diff2*diff2);
	}
}


double getFocusScore(int* image, int w, int h)
{
	double score = 0, *cuda_score;
	int N = w*h;
	int n2 = ceil(w / 2.0) * ceil(h / 2.0);
	dim3 gridSize(32, 16);
	dim3 blockSize(32, n2/32*32*16);

	int* cuda_image;

	hipMalloc((void **)&cuda_score, sizeof(double));
	hipMemcpy(cuda_score, &score, sizeof(double), hipMemcpyHostToDevice);

	// allocate image
	hipMalloc(&cuda_image, w * h * sizeof(int));

	// copy image to cuda image
	checkCudaErrors(hipMemcpy(cuda_image, image, w*h*sizeof(int), hipMemcpyDeviceToHost));
	
	cudaEnergy <<< blockSize, gridSize >>> (cuda_image, w, h, cuda_score);

	hipMemcpy(&score, cuda_score, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(cuda_image);
	hipFree(cuda_score);

	return score;
}

bool cudaScore::FindTargets(int* intensityImage, float* scoreImage, int w_d, int h_d, int regionWidth, int regionHeight, int increment_W, int increment_H, 
							float* coMatrixTarget, int coDIMX, int coDIMY, int coDIMZ, bool bCrossEntropy)
{
	// scan test image calculate intersection value save value to image
	size_t CoocSize = coDIMX*coDIMY*coDIMZ;

	dim3 gridSize(((w_d + increment_W - 1) / increment_W), (h_d + increment_H - 1) / increment_H);
	dim3 blockSize(regionWidth, regionHeight);
	//size_t regionSizeBytes = regionWidth*regionHeight * sizeof(int);
	size_t CoocSizeBytes = CoocSize * sizeof(float);
	size_t sharedMemBytes = CoocSizeBytes;

	int devID =0;
	int count=0, *dev_count;
	hipMalloc((void **)&dev_count, sizeof(int));
	hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	int N = w_d*h_d;

	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		DBOUT("hipGetDevice returned error " << hipGetErrorString(error) << " :" << error << " line=" << __LINE__ << std::endl);
		return false;
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		DBOUT("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		return false;
	}

	if (error != hipSuccess)
	{
		DBOUT("hipGetDeviceProperties returned error " << hipGetErrorString(error) << ", code=" << error << " line=" << __LINE__ << std::endl);
		return false;
	}
	else
	{
		DBOUT("GPU Device " << devID << " - " << deviceProp.name << " with compute capability major=" << deviceProp.major << " minor=" << deviceProp.minor << std::endl);

		const int kb = 1024;
		DBOUT("  Shared memory: "<< (int)deviceProp.sharedMemPerBlock / kb << " MB" << std::endl);
		DBOUT("  Constant memory: "<< (int)deviceProp.totalConstMem / kb << " MB" << std::endl);
		DBOUT("  Block registers: "<< deviceProp.regsPerBlock << std::endl);
		DBOUT("  Warp size:        "<< deviceProp.warpSize << std::endl);
		DBOUT("  Threads per block: "<< deviceProp.maxThreadsPerBlock << std::endl);
		DBOUT("  Max block dimensions: "<< deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << std::endl);
		DBOUT("  Max grid dimensions:  "<< deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << std::endl);

		DBOUT("Requested resources: " << "gridSize.x=" << gridSize.x << " gridSize.y=" << gridSize.y << " blockSize.x="<< blockSize.x <<" blockSize.y="<<blockSize.y << " sharedMemory="<< (int)sharedMemBytes / kb << " MB" << std::endl);
	}

	//checkCudaErrors(hipMemcpy(cuda_intensity_image, intensityImage, w_d * h_d * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cuda_target_cooc, coMatrixTarget, CoocSize * sizeof(float), hipMemcpyHostToDevice));

	mycudaMemsetFloat << < (N + 255) / 256, 256 >> > (cuda_score_image, N);
	mycudaMemsetInt << < (N + 255) / 256, 256 >> > (cuda_score_count_image, N);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8*sizeof(int), 0, 0, 0, hipChannelFormatKindSigned);

	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, w_d, h_d);

	checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, intensityImage,  N*sizeof(int), hipMemcpyHostToDevice));

	cuda_texture_image.addressMode[0] = hipAddressModeClamp;
	cuda_texture_image.addressMode[1] = hipAddressModeClamp;
	cuda_texture_image.filterMode = hipFilterModePoint;
	cuda_texture_image.normalized = false;

	// bind texture to image
	checkCudaErrors(hipBindTextureToArray(cuda_texture_image, cuArray, channelDesc));

	scoreTargetsGPU << < gridSize, blockSize, sharedMemBytes >> > (w_d, h_d, regionWidth, regionHeight, increment_W, increment_H,
		cuda_target_cooc, cuda_score_image, cuda_score_count_image, coDIMX, coDIMY, coDIMZ, dev_count, bCrossEntropy); // gpu

	//hipDestroyTextureObject(cuda_texture_image);
	//hipFreeArray(cuArray);

	hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

	hipFreeArray(cuArray);
	hipFree(dev_count);

	if (count != gridSize.x*gridSize.y*blockSize.x*blockSize.y)
	{
		DBOUT("ERROR: total number of threads executed: " << count << " != " << gridSize.x*gridSize.y*blockSize.x*blockSize.y << std::endl);
		return false;
	}
	else
	{
		DBOUT("Score image created sucessfully"<< std::endl);

		checkCudaErrors(hipPeekAtLastError());

		float* scoreImageTemp = new float[w_d*h_d];
		int* scoreCountImage = new int[w_d*h_d];

		checkCudaErrors(hipMemcpy(scoreImageTemp, cuda_score_image, w_d*h_d * sizeof(float), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(scoreCountImage, cuda_score_count_image, w_d*h_d * sizeof(int), hipMemcpyDeviceToHost));

		for (int i = 0; i < w_d*h_d; i++)
			scoreImage[i] = scoreImageTemp[i] / ((float)scoreCountImage[i]);

		delete[] scoreImageTemp;
	}
	
	return true;
}




