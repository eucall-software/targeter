#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <stdlib.h>
#include "math.h"
#include "targetDetectionGPU.h"
#include "globals.h"

/*
texture<int, hipTextureType2D, hipReadModeElementType> cuda_texture_image;
*/

using namespace std;

const double sqrt_2 = 1.4142135;

__global__ void haar_horizontal(float input[], float output[], int o_width, int w)
{
	int x_index = blockIdx.x*blockDim.x + threadIdx.x;
	int y_index = blockIdx.y*blockDim.y + threadIdx.y;

	if (x_index >= (w + 1) / 2 || y_index >= w) return;

	int i_thread_id = y_index*o_width + 2 * x_index;
	int o_thread_id = y_index*o_width + x_index;

	const double sqrt_2 = 1.4142135;
	output[o_thread_id] = (input[i_thread_id] + input[i_thread_id + 1]) / sqrt_2;
	output[o_thread_id + w / 2] = (input[i_thread_id] - input[i_thread_id + 1]) / sqrt_2;
}

__global__ void haar_vertical(float input[], float output[], int o_width, int w)
{
	int x_index = blockIdx.x*blockDim.x + threadIdx.x;
	int y_index = blockIdx.y*blockDim.y + threadIdx.y;

	if (y_index >= (w + 1) / 2 || x_index >= w) return;

	int p1 = 2 * y_index*o_width + x_index;
	int p2 = (2 * y_index + 1)*o_width + x_index;
	int p3 = y_index*o_width + x_index;

	const double sqrt_2 = 1.4142135;
	output[p3] = (input[p1] + input[p2]) / sqrt_2;
	output[p3 + o_width*w / 2] = (input[p1] - input[p2]) / sqrt_2;
}

void cudaScore::haar(float input[], float output[], int o_width, int o_height, int levels)
{
	float* d_input;
	float* d_output;

	int widthstep = o_width * sizeof(float);

	hipMalloc(&d_input, widthstep*o_height);
	hipMalloc(&d_output, widthstep*o_height);

	hipMemcpy(d_input, input, widthstep*o_height, hipMemcpyHostToDevice);

	dim3 blocksize(16, 16);
	dim3 gridsize;

	int w = o_width;
	gridsize.x = (w + blocksize.x - 1) / blocksize.x;
	gridsize.y = (w + blocksize.y - 1) / blocksize.y;

	for(int ct = 0; ct<levels; ct++)
	{
		haar_horizontal << <gridsize, blocksize >> >(d_input, d_output, o_width, w);
		haar_vertical << <gridsize, blocksize >> >(d_output, d_input, o_width, w);
		w /= 2;
	}

	hipMemcpy(output, d_input, widthstep*o_height, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);
}

// compute Haar transform image is square of dimensions size (with power 2)
void cudaScore::haar2d_gpu(float* input, int size, int levels)
{
	int w = size;
	float* output = new float[size*size];

	haar(input, output, w, w, levels);
}


bool Check_CUDA_Device()
{
	int devID = 0;
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	error = hipGetDeviceProperties(&deviceProp, devID);

	int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		DBOUT("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
		DBOUT("\nFAILED\n");
	}

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		DBOUT("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		return false;
	}
	else
	{
		DBOUT("GPU Device " << devID << " - " << deviceProp.name << " with compute capability major=" << deviceProp.major << " minor=" << deviceProp.minor << std::endl);

		const int kb = 1024;
		DBOUT("  Shared memory: " << (int)deviceProp.sharedMemPerBlock / kb << " MB" << std::endl);
		DBOUT("  Constant memory: " << (int)deviceProp.totalConstMem / kb << " MB" << std::endl);
		DBOUT("  Block registers: " << deviceProp.regsPerBlock << std::endl);
		DBOUT("  Warp size:        " << deviceProp.warpSize << std::endl);
		DBOUT("  Threads per block: " << deviceProp.maxThreadsPerBlock << std::endl);
		DBOUT("  Max block dimensions: " << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << std::endl);
		DBOUT("  Max grid dimensions:  " << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << std::endl);

		int driverVersion = 0, runtimeVersion = 0;

		if (devID == 0) {
			// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
			if (deviceProp.major == 9999 && deviceProp.minor == 9999)
				printf("There is no device supporting CUDA.\n");
			else if (deviceCount == 1)
				printf("There is 1 device supporting CUDA\n");
			else
				printf("There are %d devices supporting CUDA\n", deviceCount);
		}

		printf("\nDevice %d: \"%s\"\n", devID, deviceProp.name);

		typedef struct
		{
			int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
			int Cores;
		} sSMtoCores;

		sSMtoCores nGpuArchCoresPerSM[] =
		{
			{ 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
			{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
			{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
			{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
			{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
			{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
			{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
			{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
			{ -1, -1 }
		};


#if CUDART_VERSION >= 2020
		// Console log
		hipDriverGetVersion(&driverVersion);
		printf("  CUDA Driver Version:                           %d.%d\n", driverVersion / 1000, driverVersion % 100);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Runtime Version:                          %d.%d\n", runtimeVersion / 1000, runtimeVersion % 100);
#endif
		printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
		printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);

		printf("  Total amount of global memory:                 %u bytes\n", deviceProp.totalGlobalMem);
#if CUDART_VERSION >= 2000
		printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
		printf("  Number of cores:                               %d\n", nGpuArchCoresPerSM[deviceProp.major].SM * deviceProp.multiProcessorCount);
#endif
		printf("  Total amount of constant memory:               %u bytes\n", deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %u bytes\n", deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		printf("  Warp size:                                     %d\n", deviceProp.warpSize);
		printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %u bytes\n", deviceProp.memPitch);
		printf("  Texture alignment:                             %u bytes\n", deviceProp.textureAlignment);
		printf("  Clock rate:                                    %.2f GHz\n", deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >= 2000
		printf("  Concurrent copy and execution:                 %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
#endif
#if CUDART_VERSION >= 2020
		printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated:                                    %s\n", deviceProp.integrated ? "Yes" : "No");
		printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
		printf("  Compute mode:                                  %s\n", deviceProp.computeMode == hipComputeModeDefault ?
			"Default (multiple host threads can use this device simultaneously)" :
			deviceProp.computeMode == hipComputeModeExclusive ?
			"Exclusive (only one host thread at a time can use this device)" :
			deviceProp.computeMode == hipComputeModeProhibited ?
			"Prohibited (no host thread can use this device)" :
			"Unknown");
#endif
#if CUDART_VERSION >= 3000
		printf("  Concurrent kernel execution:                   %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
#endif
#if CUDART_VERSION >= 3010
		printf("  Device has ECC support enabled:                %s\n", deviceProp.ECCEnabled ? "Yes" : "No");
#endif
		// csv masterlog info
		// *****************************
		// exe and CUDA driver name 
		printf("\n");
		std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
		char cTemp[10];

		// driver version
		sProfileString += ", CUDA Driver Version = ";

		sprintf(cTemp, "%d.%d", driverVersion / 1000, driverVersion % 100);

		sProfileString += cTemp;

		// Runtime version
		sProfileString += ", CUDA Runtime Version = ";

		sprintf(cTemp, "%d.%d", runtimeVersion / 1000, runtimeVersion % 100);

		sProfileString += cTemp;

		// Device count      
		sProfileString += ", NumDevs = ";

		sprintf(cTemp, "%d", deviceCount);

		sProfileString += cTemp;

		// First 2 device names, if any
		for (devID = 0; devID < ((deviceCount > 2) ? 2 : deviceCount); ++devID)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, devID);
			sProfileString += ", Device = ";
			sProfileString += deviceProp.name;
		}
		sProfileString += "\n";

		printf("%s", sProfileString.c_str());
	}
	return true;
}

#define checkCudaErrors(err) { __checkCudaErrors((err), __FILE__, __LINE__); }
inline void __checkCudaErrors(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		const char* es = hipGetErrorString(err);

		DBOUT("file=" << (int)file << " line= "  << line << " CUDA Runtime API error (" << (int)err << ")" <<  es << std::endl);
	}
}

cudaScore::cudaScore(int w_d, int h_d, int CoocSize)
{
	// allocate image
	hipMalloc(&cuda_intensity_image, w_d * h_d * sizeof(int));

	// allocate target image
	hipMalloc(&cuda_target_cooc, CoocSize * sizeof(float));

	// allocate score image
	hipMalloc(&cuda_score_image, w_d * h_d * sizeof(float));

	hipMalloc(&cuda_score_count_image, w_d * h_d * sizeof(int));
}

cudaScore::~cudaScore()
{
	hipFree(cuda_intensity_image);
	hipFree(cuda_target_cooc);
	hipFree(cuda_score_image);
	hipFree(cuda_score_count_image);
}


__global__
void mycudaMemsetInt(int* image, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		image[i] = 0;
	}
}

__global__
void mycudaMemsetFloat(float* image, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
	{
		image[i] = 0.0;
	}
}

/**
*
*  Gets coocurrance matrix for image region for grayscale image
*
* @author    David Watts
* @since     2017/03/07
*
* FullName   getCoocMatrixGrayGPU
* Qualifier
* @param     int* & m
* @param     int * pMask
* @param     int startX
* @param     int startY
* @param     int regionWidth
* @param     int regionHeight
* @param     float * coMatrixF
* @param     int NoClusters
* @param     int maxD
* @return    float
* Access     public
*/
__device__
void getCoocMatrixGrayGPU(int* pImage, int image_ind_x, int image_ind_y, int width, int height,
						int regionWidth, int regionHeight, float* coMatrixRegion, int averageCluster,
						int maxDist, int coDIMX, int coDIMY, int coDIMZ, bool bFASTCOOC)	
{
	int local_sum = 0;
	int pixel_index = image_ind_x + image_ind_y*width;
	
	if (threadIdx.x < regionWidth && threadIdx.y < regionHeight)
	{
		int c1 = pImage[image_ind_x + image_ind_y * width]; //tex2D(cuda_texture_image, image_ind_x, image_ind_y); 

		if (c1 >= 0)
		{
			if(bFASTCOOC)
			{
				float dx = (threadIdx.x);
				float dy = (threadIdx.y);

				float dxy = sqrtf(dx*dx+dy*dy);
				int d = MIN(coDIMY-1, int(coDIMY*(dxy/maxDist)));

				int dC = MIN(coDIMX-1, MAX(0, coDIMX/2 + (averageCluster - c1)));	// offset to handle negative numbers

				int ind = dC + coDIMX * d;	// 2D indexing

				if (ind < coDIMX*coDIMY)
				{
					atomicAdd(&coMatrixRegion[ind], 1.0);
					atomicAdd(&coMatrixRegion[0], 1.0);	
				}
			}
			else
			{
				int nextIndex = 1 + threadIdx.x + threadIdx.y*regionWidth;

				int nextJ = nextIndex / regionWidth;
				int nextI = nextIndex % regionWidth;

				//printf("rw=%d rh=%d\n", regionWidth, regionHeight);

				// check against all others - the slow part !!!
				for (int y = nextJ; y < regionHeight; y++)
					for (int x = nextI; x < regionWidth; x++)
					{
						if (image_ind_x + x < width && image_ind_y + y < height)
						{
							int c2 = pImage[(image_ind_x + x) + (image_ind_y + y)*width];  //tex2D(cuda_texture_image, image_ind_x + x, image_ind_y + y);

							if (c2 >= 0)
							{
								float dx = (x - threadIdx.x);
								float dy = (y - threadIdx.y);

								float dxy = sqrtf(dx*dx+dy*dy);
								int d = MIN(coDIMZ-1, int(coDIMZ*(dxy/maxDist)));

								int ind = c1 + coDIMX * (c2 + coDIMY * d);

								if (ind < coDIMX*coDIMY*coDIMZ)
								{
									atomicAdd(&coMatrixRegion[ind], 1.0);
									atomicAdd(&coMatrixRegion[0], 1.0);	
								}
							}
						}
					}
			}

			
		}
	}
}

/**
*
*  Gets target score image based on matching image regions with training coccurrance mattrix (coMatrixTestF)
*
* @author    David Watts
* @since     2017/03/07
*
* FullName   findTargets
* Qualifier
* @param     int* & detectionImage
* @param     float * coMatrixTestF
* @param     int NoClusters
* @param     int maxD
* @return    float*
* Access     public
*/
__global__
void scoreTargetsGPU(int* cuda_intensity_image, int w, int h, int regionWidth, int regionHeight, int increment_W, int increment_H,
					float* coMatrixTarget, float* scoreImage, int* countImage, float maxDist, int coDIMX, int coDIMY, 
					int coDIMZ, int averageCluster, int* ct, bool bCrossEntropy, bool bFASTCOOC)
{
	__shared__ float shared_score;

	extern __shared__ float sh[];
	float* sharedCoocRegion = sh;
	//int* sharedCoocRegion = (int*)sh + regionWidth*regionHeight;

	// index into image and sub tile
	int image_ind_x = blockIdx.x*increment_W + threadIdx.x;
	int image_ind_y = blockIdx.y*increment_H + threadIdx.y;

	// zero shared memory
	int tid = threadIdx.y*blockDim.x + threadIdx.x;			// get linear thread index
	int cN = coDIMX*coDIMY*coDIMZ;

	if(bFASTCOOC)
		cN = coDIMX*coDIMY;
	
	int blockSize = blockDim.x * blockDim.y;

	// zero co-occurrence matrix using parallel threads, and loop to fill in rest up to cN
	for(int i=tid; i<cN; i += blockSize)
		sharedCoocRegion[i] = 0;
		
	__syncthreads();	// all threads in block synchronised
/*
	// copy image region into into shared tile memory
	if (image_ind_x < w && image_ind_y < h)
		sharedTile[threadIdx.x + threadIdx.y*regionWidth] = tex2D(cuda_texture_image, image_ind_x, image_ind_y);
	else
		sharedTile[threadIdx.x + threadIdx.y*regionWidth] = -1;
*/
	
	// create the coocurrence matrices

	// last data point has nothing to check against
	if (threadIdx.x != regionHeight - 1 && threadIdx.y != regionHeight - 1)
	{
		// takes pixel value compares with neighbours
		getCoocMatrixGrayGPU(cuda_intensity_image, image_ind_x, image_ind_y, w, h, regionWidth, regionHeight, sharedCoocRegion, 
							averageCluster, maxDist, coDIMX, coDIMY, coDIMZ, bFASTCOOC);
	}

	__syncthreads();

	// loop though coocurance matrices and get score
	float local_score = 0.0;
		
	// in the case of sharedCooc having bigger dimensions than the block size
	// zero remaining part of memory
	if (tid == 0)
		shared_score = 0.0;
	
	for (int i = tid; i < cN; i += blockSize)
	{
		if(i != 0)
			local_score += fmin(coMatrixTarget[i], sharedCoocRegion[i]/sharedCoocRegion[0]);	// minimum intersection
	}

	if (local_score>0)
		atomicAdd(&shared_score, local_score);

	__syncthreads();

	// label image region with score

	if (image_ind_x < w && image_ind_y < h)
	{
		atomicAdd(&scoreImage[image_ind_x + image_ind_y*w], shared_score);

		//scoreImage[image_ind_x + image_ind_y*w] = threadIdx.x;

		atomicAdd(&countImage[image_ind_x + image_ind_y*w], 1);

		//printf("%f ", scoreImage[image_ind_x + image_ind_y*w]);
	}
	
	//////////////////////////////// sum check /////////////////////////////////////
	atomicAdd(ct, 1);
}

/*

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8*sizeof(int), 0, 0, 0, hipChannelFormatKindSigned);

hipArray* cuArray;
hipMallocArray(&cuArray, &channelDesc, w_d, h_d);

checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, intensityImage,  N*sizeof(int), hipMemcpyHostToDevice));

cuda_texture_image.addressMode[0] = hipAddressModeClamp;
cuda_texture_image.addressMode[1] = hipAddressModeClamp;
cuda_texture_image.filterMode = hipFilterModePoint;
cuda_texture_image.normalized = false;

// bind texture to image
checkCudaErrors(hipBindTextureToArray(cuda_texture_image, cuArray, channelDesc));
*/

bool cudaScore::setupCudaDevice(hipDeviceProp_t* deviceProp, int &devID, dim3 gridSize, dim3 blockSize, size_t sharedMemBytes)
{
	hipError_t error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		DBOUT("hipGetDevice returned error " << hipGetErrorString(error) << " :" << error << " line=" << __LINE__ << std::endl);
		return false;
	}

	error = hipGetDeviceProperties(deviceProp, devID);

	if (error != hipSuccess)
	{
		DBOUT("hipGetDeviceProperties returned error " << hipGetErrorString(error) << " :" << error << " line=" << __LINE__ << std::endl);
		return false;
	}
	
	if (deviceProp->computeMode == hipComputeModeProhibited)
	{
		DBOUT("Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		return false;
	}

	DBOUT("GPU Device " << devID << " - " << deviceProp->name << " with compute capability major=" << deviceProp->major << " minor=" << deviceProp->minor << std::endl);

	const int kb = 1024;
	DBOUT("  Shared memory per block: " << (int)deviceProp->sharedMemPerBlock / kb << " MB , " << (int)deviceProp->sharedMemPerBlock << " bytes" << std::endl);
	DBOUT("  Constant memory: " << (int)deviceProp->totalConstMem / kb << " MB" << std::endl);
	DBOUT("  Registers per Block: " << deviceProp->regsPerBlock << std::endl);
	DBOUT("  Warp size:        " << deviceProp->warpSize << std::endl);
	DBOUT("  Maximum Threads per block: " << deviceProp->maxThreadsPerBlock << std::endl);
	DBOUT("  Max block dimensions: " << deviceProp->maxThreadsDim[0] << ", " << deviceProp->maxThreadsDim[1] << ", " << deviceProp->maxThreadsDim[2] << std::endl);
	DBOUT("  Max grid dimensions:  " << deviceProp->maxGridSize[0] << ", " << deviceProp->maxGridSize[1] << ", " << deviceProp->maxGridSize[2] << std::endl);

	DBOUT("Requested resources: " << "gridSize.x=" << gridSize.x << " gridSize.y=" << gridSize.y << " blockSize.x=" << blockSize.x << " blockSize.y=" << blockSize.y
		<< " sharedMemory=" << (int)sharedMemBytes / kb << " MB, " << (int)sharedMemBytes << " bytes" << std::endl);

	return true;
}

bool cudaScore::FindTargets(int* detectionImage, float* scoreImage, int width, int height, COOCMatrix* coocTraining,
							bool bIntensityImage, bool bCrossEntropy, bool bFASTCOOC)
{
	// scan test image calculate intersection value save value to image
	size_t CoocSize = coocTraining->coDIMX*coocTraining->coDIMY*coocTraining->coDIMZ;

	if(bFASTCOOC)
		CoocSize = coocTraining->coDIMX*coocTraining->coDIMY;	// 2D matrix

	dim3 gridSize(((width + coocTraining->incrementWidth - 1) / coocTraining->incrementWidth), (height + coocTraining->incrementHeight - 1) / coocTraining->incrementHeight);
	dim3 blockSize(coocTraining->regionWidth, coocTraining->regionHeight);
	//size_t regionSizeBytes = regionWidth*regionHeight * sizeof(int);
	size_t CoocSizeBytes = CoocSize * sizeof(float);
	size_t sharedMemBytes = CoocSizeBytes;

	int devID =0;
	int count=0, *dev_count;
	hipMalloc((void **)&dev_count, sizeof(int));
	hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	int N = width*height;

	if (!setupCudaDevice(&deviceProp, devID, gridSize, blockSize, sharedMemBytes))
		return false;

	//checkCudaErrors(hipMemcpy(cuda_intensity_image, intensityImage, w_d * h_d * sizeof(int), hipMemcpyHostToDevice));
	if(bIntensityImage) {
		checkCudaErrors(hipMemcpy(cuda_target_cooc, coocTraining->coMatrixIntensity, CoocSize * sizeof(float), hipMemcpyHostToDevice));
	}
	else {
		checkCudaErrors(hipMemcpy(cuda_target_cooc, coocTraining->coMatrixHue, CoocSize * sizeof(float), hipMemcpyHostToDevice));
	}

	mycudaMemsetFloat << < (N + 255) / 256, 256 >> > (cuda_score_image, N);
	mycudaMemsetInt << < (N + 255) / 256, 256 >> > (cuda_score_count_image, N);

	checkCudaErrors(hipMemcpy(cuda_intensity_image, detectionImage, width * height * sizeof(int), hipMemcpyHostToDevice));

	float maxDist = sqrtf(coocTraining->regionWidth*coocTraining->regionWidth + coocTraining->regionHeight*coocTraining->regionHeight);

	scoreTargetsGPU <<< gridSize, blockSize, sharedMemBytes >>>(cuda_intensity_image, width, height, 
																coocTraining->regionWidth, coocTraining->regionHeight, 
																coocTraining->incrementWidth, coocTraining->incrementHeight,
																cuda_target_cooc, cuda_score_image, cuda_score_count_image, maxDist, 
																coocTraining->coDIMX, coocTraining->coDIMY, coocTraining->coDIMZ, 
																coocTraining->averageIntensity, dev_count, bCrossEntropy, bFASTCOOC); // gpu

	//hipDestroyTextureObject(cuda_texture_image);
	//hipFreeArray(cuArray);

	hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_count);

	if (count != gridSize.x*gridSize.y*blockSize.x*blockSize.y)
	{
		DBOUT("ERROR: total number of threads executed: " << count << " != " << gridSize.x*gridSize.y*blockSize.x*blockSize.y << std::endl);
		return false;
	}
	else
	{
		DBOUT("Score image created sucessfully"<< std::endl);

		checkCudaErrors(hipPeekAtLastError());

		float* scoreImageTemp = new float[width*height];
		int* scoreCountImage = new int[width*height];

		checkCudaErrors(hipMemcpy(scoreImageTemp, cuda_score_image, width*height*sizeof(float), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(scoreCountImage, cuda_score_count_image, width*height*sizeof(int), hipMemcpyDeviceToHost));

		for (int i = 0; i < width*height; i++)
			scoreImage[i] = scoreImageTemp[i] / ((float)scoreCountImage[i]);

		delete[] scoreImageTemp;
	}
	
	return true;
}

/*
cv::Mat cudaScore::FindTargets::detectLawsTextureFeatures(cv::Mat detectionImage, QVector<cv::Mat> lawsHistTarget, QVector<float> biases)
{
	int regionSize = 10;
	int histRange = 128;
	int histSize = 10;

	// now get detection 
	cv::Mat im, scoreImage(detectionImage.rows, detectionImage.cols, CV_32FC1);

	// now filter detect image and compare histograms of regions of the image
	cv::cvtColor(detectionImage, im, CV_BGR2GRAY, 1);

	cv::Ptr<cv::CLAHE> clahe = cv::createCLAHE(2.0);
	clahe->apply(im, im);

	im.convertTo(im, CV_32FC1);

	QMap<QString, cv::Mat> lawsMapDetect = getLawFilteredImages(im);

	QVector<cv::Mat> lawsHistDetect;

	for (int c = 0; c < 8; c++)
		lawsHistDetect.append(cv::Mat());

	// but this has to be on regions of the detection image
	for (int i = 0; i < im.cols; i += regionSize)
		for (int j = 0; j < im.rows; j += regionSize)
		{
			int regionSizeI = regionSize;
			int regionSizeJ = regionSize;

			if (i + regionSize >= im.cols)
				regionSizeI = im.cols - i;
			if (j + regionSize >= im.rows)
				regionSizeJ = im.rows - j;

			// get histograms in this region of the image
			cv::Rect roi(i, j, regionSizeI, regionSizeJ);

			addHistogram(lawsHistDetect[0], lawsMapDetect["im_SS"](roi), cv::Mat(), cv::Mat(), cv::Mat(), histSize, histRange, false);
			addHistogram(lawsHistDetect[1], lawsMapDetect["im_RR"](roi), cv::Mat(), cv::Mat(), cv::Mat(), histSize, histRange, false);
			addHistogram(lawsHistDetect[2], lawsMapDetect["im_LS"](roi), lawsMapDetect["im_SL"](roi), cv::Mat(), cv::Mat(), histSize, histRange, false);
			addHistogram(lawsHistDetect[3], lawsMapDetect["im_SR"](roi), lawsMapDetect["im_RS"](roi), cv::Mat(), cv::Mat(), histSize, histRange, false);
			addHistogram(lawsHistDetect[4], lawsMapDetect["im_EE"](roi), lawsMapDetect["im_EER"](roi), cv::Mat(), cv::Mat(), histSize, histRange, false);
			addHistogram(lawsHistDetect[5], lawsMapDetect["im_EL"](roi), lawsMapDetect["im_LE"](roi), lawsMapDetect["im_LER"](roi), lawsMapDetect["im_ERL"](roi), histSize, histRange, false);
			addHistogram(lawsHistDetect[6], lawsMapDetect["im_ES"](roi), lawsMapDetect["im_SE"](roi), lawsMapDetect["im_SER"](roi), lawsMapDetect["im_ERS"](roi), histSize, histRange, false);
			addHistogram(lawsHistDetect[7], lawsMapDetect["im_ER"](roi), lawsMapDetect["im_RE"](roi), lawsMapDetect["im_RER"](roi), lawsMapDetect["im_ERR"](roi), histSize, histRange, false);

			//score detect region using target histogram	
			float s = scoreLawsHistogram(lawsHistDetect, lawsHistTarget, biases);

			for (int k = i; k < i + regionSizeI; k++)
				for (int l = j; l < j + regionSizeJ; l++)
				{
					scoreImage.at<float>(l, k) = s;
				}
		}

	// create greyscale image of score
	cv::Mat sim = HelperFunctions::convertFloatToGreyscaleMat(scoreImage);

	return sim;
}
*/