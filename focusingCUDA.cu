#include "hip/hip_runtime.h"
#include <iomanip>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "math.h"
#include "focusingCUDA.h"

#define checkCudaErrors(err) { __checkCudaErrors((err), __FILE__, __LINE__); }
inline void __checkCudaErrors(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		const char* es = hipGetErrorString(err);

		DBOUT("file=" << (int)file << " line= " << line << " CUDA Runtime API error (" << (int)err << ")" << es << std::endl);
	}
}

__global__
void cudaEnergy(int* image, int w, int h, int* dev_score)
{
	uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint j = (blockIdx.y * blockDim.y) + threadIdx.y;

	//printf("i=%d j=%d v=", i, j);

	// Test to see if we're testing a valid pixel
	if (i < w && j < h)
	{
		int diff1 = 0, diff2 = 0;

		int val1 = image[i + j*w];

		printf("%d\n", val1);

		if ((i + 1) < w)
			diff1 = val1 - image[(i + 1) + j*w];

		if ((j + 1) < h)
			diff2 = val1 - image[i + (j + 1)*w];

		int ssq = abs(diff1) + abs(diff2);	// rms

		atomicAdd(dev_score, ssq);
	}
}

int* getCImage(const cv::Mat& img) // returns true if the given 3 channel image is B = G = R
{
	int* im = new int[img.rows*img.cols];

	for (int j = 0; j < img.rows; j++)
	{
		for (int i = 0; i < img.cols; i++)
		{
			cv::Vec3b val = img.at<cv::Vec3b>(j, i);

			im[i + j*img.cols] = (int)val[0] * 0.114 + val[1] * 0.587 + val[2] * 0.299;
		}
	}
	return im;
}

void focusingCUDA::getFocusScore(FocusResult& res)
{
	int score = 0, *cuda_score;
	
	int w = res.focusImage.cols, int h = res.focusImage.rows;
	int N = w*h;

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(w/threadsPerBlock.x, h/threadsPerBlock.y);

	int* image = getCImage(res.focusImage);
	int* cuda_image;

	hipMalloc((void **)&cuda_score, sizeof(int));
	checkCudaErrors(hipMemcpy(cuda_score, &score, sizeof(int), hipMemcpyHostToDevice));

	// allocate image
	hipMalloc(&cuda_image, w * h * sizeof(int));

	// copy image to cuda image
	checkCudaErrors(hipMemcpy(cuda_image, image, w * h * sizeof(int), hipMemcpyHostToDevice));

	printf("in cuda function\n");

	// call cuda function
	cudaEnergy << < numBlocks, threadsPerBlock >> > (cuda_image, w, h, cuda_score);

	checkCudaErrors(hipMemcpy(&score, cuda_score, sizeof(int), hipMemcpyDeviceToHost));

	hipFree(cuda_image);
	hipFree(cuda_score);

	if(image != nullptr)
		delete[] image;

	res.focusValue = score;

	//emit setFOCUS(res);
}

